#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime.h"
#include "hiprand/hiprand_kernel.h"
#include "thrust/device_vector.h"
#include "thrust/host_vector.h"
#include "thrust/device_ptr.h"
#include <thrust/sort.h>
#include <thrust/execution_policy.h>
#include <thrust/sequence.h>
#include <ctime>
#include <stdio.h>
#include <iostream>
#include <math.h>
#include <iomanip>
#include <vector>
#include <stdlib.h>
#include <fstream>
#include <string>
#include "my_functions.h"


__global__ void calc_out_kernel(Snake_GPU* player)
{

	hiprandState state;
	hiprand_init((unsigned long long)clock(), 0, 0, &state);
	player->local_state = state;
	player->look_around();
	int out = player->calculate_output();
	player->move(out);
}

#if EVOLVE_ON_GPU

__global__ void setup_cuRand(hiprandState* state)
{
	int id = threadIdx.x + blockDim.x + blockIdx.x;
	if (id < POP_SIZE)
		hiprand_init(1234, id, 0, &state[id]);
}

__global__ void evaluate_fintess_kernell(Snake_GPU* dev_snakes, float* dev_fit, hiprandState* st, int init_cur)
{
	int threadID = threadIdx.x + blockDim.x * blockIdx.x;
	int krok = blockDim.x * gridDim.x;
	if (threadID < POP_SIZE && init_cur == 0)
	{
		dev_snakes[threadID].local_state = st[threadID];
	}
	for (int i = threadID; i < POP_SIZE; i += krok)
	{
		dev_snakes[i].play_game();
		dev_fit[i] = dev_snakes[i].fitness;
	}
}

__global__ void crossover_kernel(Snake_GPU* dev_snakes, Snake_GPU* best_snakes, float* dev_fit, hiprandState* st)
{
	int threadID_x = threadIdx.x + blockDim.x * blockIdx.x;
	
	if (threadID_x < POP_SIZE)
	{
		int krok_x = blockDim.x * gridDim.x;

		if (threadID_x == 0)
		{
			printf("%f;%d;", dev_fit[0], best_snakes[0].score);
		}

		int j = threadID_x;
		hiprandState state;
		hiprand_init(1234, threadID_x, 0, &state);

			dev_snakes[j].reset();
			int first = hiprand(&state) % BEST_SNAKES_SIZE;
			int second = hiprand(&state) % (BEST_SNAKES_SIZE - 1);

			//printf("which: %d || FIRST: %d || SECOND: %d \n",j, first, second);

			//WAGI
			for (int i = 0; i < INPUT_WEIGHTS_SIZE; i += 1)
			{
				if (hiprand(&state) % 2 == 0)
					dev_snakes[j].input_weights[i] = best_snakes[first].input_weights[i];
				else
					dev_snakes[j].input_weights[i] = best_snakes[second].input_weights[i];
			}
			for (int i = 0; i < HIDDEN_WEGHTS_SIZE; i += 1)
			{
				if (hiprand(&state) % 2 == 0)
					dev_snakes[j].hidden_weights[i] = best_snakes[first].hidden_weights[i];
				else
					dev_snakes[j].hidden_weights[i] = best_snakes[second].hidden_weights[i];
			}

			for (int i = 0; i < OUTPUT_WEIGHTS_SIZE; i += 1)
			{
				if (hiprand(&state) % 2 == 0)
					dev_snakes[j].output_weights[i] = best_snakes[first].output_weights[i];
				else
					dev_snakes[j].output_weights[i] = best_snakes[second].output_weights[i];
			}

		
			for (int i = 0; i < HIDDEN_NEURONS_SIZE; i += 1)
			{
				if (hiprand(&state) % 2 == 0)
					dev_snakes[j].hidden_bias_0[i] = best_snakes[first].hidden_bias_0[i];
				else
					dev_snakes[j].hidden_bias_0[i] = best_snakes[second].hidden_bias_0[i];
			}
			for (int i = 0; i < HIDDEN_NEURONS_1_SIZE; i += 1)
			{
				if (hiprand(&state) % 2 == 0)
					dev_snakes[j].hidden_bias_1[i] = best_snakes[first].hidden_bias_1[i];
				else
					dev_snakes[j].hidden_bias_1[i] = best_snakes[second].hidden_bias_1[i];
			}

			for (int i = 0; i < OUTPUT_NEURONS_SIZE; i += 1)
			{
				if (hiprand(&state) % 2 == 0)
					dev_snakes[j].output_bias[i] = best_snakes[first].output_bias[i];
				else
					dev_snakes[j].output_bias[i] = best_snakes[second].output_bias[i];
			}

			//MUTACJA
			int r = 0;
			//WAGI
			for (int i = 0; i < INPUT_WEIGHTS_SIZE; i += 1)
			{
				if (hiprand(&state) % 100 <= MUTATION_RATE)
				{
					if (hiprand(&state) % 2 == 0)
						r = 1;
					else
						r = -1;

					dev_snakes[j].input_weights[i] += MUTATION_SIZE * r;
				}
			}
			for (int i = 0; i < HIDDEN_WEGHTS_SIZE; i += 1)
			{
				if (hiprand(&state) % 100 <= MUTATION_RATE)
				{
					if (hiprand(&state) % 2 == 0)
						r = 1;
					else
						r = -1;
					dev_snakes[j].hidden_weights[i] += MUTATION_SIZE * r;
				}
			}

			for (int i = 0; i < OUTPUT_WEIGHTS_SIZE; i += 1)
			{
				if (hiprand(&state) % 100 <= MUTATION_RATE)
				{
					if (hiprand(&state) % 2 == 0)
						r = 1;
					else
						r = -1;
					dev_snakes[j].output_weights[i] += MUTATION_SIZE * r;
				}
			}

			//BIASY
			for (int i = 0; i < HIDDEN_NEURONS_SIZE; i += 1)
			{
				if (hiprand(&state) % 100 <= MUTATION_RATE)
				{
					if (hiprand(&state) % 2 == 0)
						r = 1;
					else
						r = -1;
					dev_snakes[j].hidden_bias_0[i] += MUTATION_SIZE * r;
				}
			}
			for (int i = 0; i < HIDDEN_NEURONS_1_SIZE; i += 1)
			{
				if (hiprand(&state) % 100 <= MUTATION_RATE)
				{
					if (hiprand(&state) % 2 == 0)
						r = 1;
					else
						r = -1;
					dev_snakes[j].hidden_bias_1[i] += MUTATION_SIZE * r;
				}

			}

			for (int i = 0; i < OUTPUT_NEURONS_SIZE; i += 1)
			{
				if (hiprand(&state) % 100 <= MUTATION_RATE)
				{
					if (hiprand(&state) % 2 == 0)
						r = 1;
					else
						r = -1;
					dev_snakes[j].output_bias[i] += MUTATION_SIZE * r;
				}
			}

		}
	
}

#endif

int main()
{
	bool load_from_file = true;
	std::string path_load = "SAVED/GPU_26_01_2020_12x12/GENERATION_1767.txt";

	bool use_keyboard = false;
	bool train_on_cpu = false;

	clock_t start_all_generations, end_all_generations;
	clock_t start_single_gen, end_single_gen;
	clock_t start_mut_and_cross, end_mut_and_cross;

	srand(time(NULL));

#if USE_CPU
	Snake player;

	//GRAFIKA
	sf::RenderWindow window(sf::VideoMode(WIDTH * CELL_SIZE, HEIGHT * CELL_SIZE), "SFML works!");
	sf::RenderWindow side_window(sf::VideoMode(200, 400), "SFML works!");
	side_window.setPosition(window.getPosition() + sf::Vector2i(600, 0));
	sf::Font font;
	font.loadFromFile("fonts/arial.ttf");
	sf::Text scoreText;
	scoreText.setFont(font);
	scoreText.setCharacterSize(12);
	scoreText.setFillColor(sf::Color::White);
	scoreText.setPosition(0, 0);

	if (load_from_file)
	{

		Snake_GPU host_snake = load_snake_gpu(path_load);
		Snake_GPU* dev_snake = 0;
		Snake_GPU player_gpu;
		hipMalloc(&dev_snake, sizeof(Snake_GPU));
		hipMemcpy(dev_snake, &host_snake, sizeof(Snake_GPU), hipMemcpyHostToDevice);

		while (player_gpu.DEAD == 0)
		{
			calc_out_kernel << <1, 1 >> > (dev_snake);
			hipDeviceSynchronize();

			hipMemcpy(&player_gpu, dev_snake, sizeof(Snake_GPU), hipMemcpyDeviceToHost);

			scoreText.setString("SCORE: " + std::to_string(player_gpu.score) + "\nSTEPS_TAKEN: " + std::to_string(player_gpu.steps_taken)
				+ "\nSTEPS_LEFT: " + std::to_string(player_gpu.steps_left));

			window.clear();
			side_window.clear();
			show_snake_gpu(&window, player_gpu);
			side_window.draw(scoreText);
			window.display();
			side_window.display();

		}

		/*player = load_snake(path_load);
		while (player.dead == 0)
		{
			
			player.look_around();
			int out = player.calculate_output();
			player.move(out);

			scoretext.setstring("score: " + std::to_string(player.score) + "\nsteps_taken: " + std::to_string(player.steps_taken)
				+ "\nsteps_left: " + std::to_string(player.steps_left));

			window.clear();
			side_window.clear();
			show_snake(&window, player);
			side_window.draw(scoretext);
			window.display();
			side_window.display();


		}*/
	}

	if (use_keyboard)
	{
		while (window.isOpen() && player.DEAD == 0)
		{
			sf::Event event;
			while (window.pollEvent(event))
			{
				switch (event.type)
					{
						// window closed
					case sf::Event::Closed:
						window.close();
						break;

						// key pressed
					case sf::Event::KeyPressed:
						if (event.key.code == sf::Keyboard::W)
						{
							player.dir = UP;
						}
						if (event.key.code == sf::Keyboard::D)
						{
							player.dir = RIGHT;
						}
						if (event.key.code == sf::Keyboard::S)
						{
							player.dir = DOWN;
						}
						if (event.key.code == sf::Keyboard::A)
						{
							player.dir = LEFT;
						}


						player.move(-10);

						player.look_around();
					}
				}

				window.clear();
				show_snake(&window, player);
				window.display();
			}
		}

	if (train_on_cpu)
	{
		start_all_generations = clock();
		Snake temp;
		std::vector<Snake> snakes(POP_SIZE);
		std::vector<Snake> best_snakes;

		int out = 0;
		int first = 0;
		int second = 0;

		std::ofstream logs("SAVED/CPU/LOGS.txt");
		logs << "GENERATION;BEST_FITNESS;BEST_SCORE;GENERATION_TIME" << std::endl;
		printf("GENERATION;BEST_FITNESS;BEST_SCORE;GENERATION_TIME \n");

		for (int i = 0; i < GENERATION; i++)
		{
			start_single_gen = clock();

			

			for (int i = 0; i < POP_SIZE; i++)
			{
				snakes.at(i).id = i;
				snakes.at(i).play_game();
			}

			std::sort(snakes.begin(), snakes.end(), [](const Snake& lhs, const Snake& rhs)
			{
					return lhs.fitness > rhs.fitness;
			});

			for (int i = 0; i < BEST_SNAKES_SIZE; i++)
			{
				best_snakes.push_back(snakes.at(i));
			}
			save_to_txt(best_snakes.at(0), "SAVED/CPU/GENERATION_", i);

			snakes.clear();

			start_mut_and_cross = clock();
			for (int i = 0; i < POP_SIZE; i++)
			{
				
				first = rand() % BEST_SNAKES_SIZE;
				second = rand() % (BEST_SNAKES_SIZE - 1);
				if (second >= first)
				{
					++second;
				}
				if (rand() % 100 <= CHILDESS_PARENT_PROB)
				{
					temp = childess_parent(best_snakes.at(0));
				}
				else
				{
					temp = cross_over(best_snakes.at(first), best_snakes.at(second));
					mutate(&temp);
				}
				snakes.push_back(temp);
			}
			end_mut_and_cross = clock();
			end_single_gen = clock();
			double czas = measure_and_show_time(start_single_gen, end_single_gen, "this generation time: ");
			logs << i <<";" << best_snakes.at(0).fitness << ";"<<best_snakes.at(0).score << ";" << czas<< std::endl;
			printf("%d;%f;%d;%f \n",i, best_snakes.at(0).fitness, best_snakes.at(0).score, czas);

			best_snakes.clear();

			
		}
		logs.close();

		end_all_generations = clock();
		measure_and_show_time(start_all_generations, end_all_generations, "all simulation time: ");
	}

#endif
		
#if EVOLVE_ON_GPU
			start_all_generations = clock();

			std::ofstream logs("SAVED/GPU/LOGS.txt");
			thrust::host_vector<Snake_GPU> h_snakes(POP_SIZE);
			thrust::host_vector<Snake_GPU> h_best_snakes(BEST_SNAKES_SIZE);

			thrust::device_vector<int> ids(POP_SIZE);
			thrust::device_vector<float> fitnesses(POP_SIZE);


			int block_size = 512;
			int num_blocks = (POP_SIZE + block_size - 1) / block_size;
			hiprandState* randState;
			int myCurandSize = POP_SIZE;
			hipMalloc((void**)&randState, myCurandSize * sizeof(hiprandState));
			setup_cuRand << <num_blocks, block_size >> > (randState);
			hipDeviceSynchronize();

			for (int i = 0; i < POP_SIZE; i++)
			{
				h_snakes[i].id = i;

			}

			thrust::device_vector<Snake_GPU> best_snakes_dev = h_best_snakes;
			thrust::device_vector<Snake_GPU> d_snakes = h_snakes;

			Snake_GPU* dev_ptr = thrust::raw_pointer_cast(&d_snakes[0]);
			float* dev_ptr_fit = thrust::raw_pointer_cast(&fitnesses[0]);
			Snake_GPU* dev_ptr_best_snakes = thrust::raw_pointer_cast(&best_snakes_dev[0]);

		
			dim3 grid(1000, 1, 1);
			dim3 block(7, 120, 1);

			logs << "GENERATION;BEST_FITNESS;BEST_SCORE;GENERATION_TIME" << std::endl;
			printf("GENERATION;BEST_FITNESS;BEST_SCORE;GENERATION_TIME \n");


			int init_cur = 0;
			for (int i = 0; i < GENERATION; i++)
			{
				start_single_gen = clock();
				printf("%d;", i);

				evaluate_fintess_kernell << < 10000, 512 >> > (dev_ptr, dev_ptr_fit, randState,init_cur);
				hipDeviceSynchronize();
				init_cur = 1;


				thrust::sequence(thrust::device, ids.begin(), ids.end(), 0);
				thrust::sort_by_key(thrust::device, fitnesses.begin(), fitnesses.end(), ids.begin(), thrust::greater<float>());
				hipDeviceSynchronize();

				for (int i = 0; i < BEST_SNAKES_SIZE; i++)
				{
					best_snakes_dev[i] = d_snakes[ids[i]];
				}


				save_to_txt_gpu(best_snakes_dev[0], "SAVED/GPU_26_01_2020_12x12/GENERATION_", i);

				start_mut_and_cross = clock();

				crossover_kernel << < 10000, 512 >> > (dev_ptr, dev_ptr_best_snakes, dev_ptr_fit,randState);
				hipDeviceSynchronize();

				end_mut_and_cross = clock();
				end_single_gen = clock();

				//measure_and_show_time(start_mut_and_cross, end_mut_and_cross, "mutation and cross time: ");
				double czas = measure_and_show_time(start_single_gen, end_single_gen, "this generation time: ");
				printf("%f \n", czas);
				//logs << i << ";" << best_snakes.at(0).fitness << ";" << best_snakes.at(0).score << ";" << czas << std::endl;
	
			}
			logs.close();
			end_all_generations = clock();
			measure_and_show_time(start_all_generations, end_all_generations, "all simulation time: ");
#endif




    return 0;
}